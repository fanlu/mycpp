#include <iostream>

#include "hip/hip_runtime.h"


//定义矢量长度
const int N = 64 * 256;

// 定义每个Block中包含的Thread数量 
const int threadsPerBlock = 256;

// 定义每个Grid中包含的Block数量, 这里32 < 64， 是为了模拟线程数量不足的情况
const int blocksPerGrid = 32;


// 核函数：矢量点积
__global__ void dot(float* a, float* b, float* c)
{
    // 声明共享内存用于存储临时乘积结果，内存大小为1个Block中的线程数量
    // PS. 每个Block都相当于有一份程序副本，因此相当于每个Block都有这样的一份共享内存
    __shared__ float cache[threadsPerBlock];

    // 线程索引
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // 一个Block中的线程索引 
    int cacheIndex = threadIdx.x;

    // 计算分量乘积，同时处理线程不足的问题
    float temp = 0.0f;
    while (tid < N)
    {
        temp += a[tid] * b[tid];
        tid  += gridDim.x * blockDim.x;
    }

    // 存储临时乘积结果
    cache[cacheIndex] = temp;

    // 对线程块中的所有线程进行同步
    // 线程块中的所有线程都执行完前面的代码后才会继续往后执行
    __syncthreads();

    // 合并算法要求长度为2的指数倍
    int i = threadsPerBlock / 2;
    while (i != 0)
    {
        if (cacheIndex < i)
            cache[cacheIndex] += cache[cacheIndex + i];
        __syncthreads();
        i /= 2;
    }

    if (cacheIndex == 0)
        c[blockIdx.x] = cache[0];
}


int main()
{
    // 在主机端创建数组
    float a[N];
    float b[N];
    float c[threadsPerBlock];
    for (size_t i = 0; i < N; i++)
    {
        a[i] = 1.f;
        b[i] = 1.f;
    }

    // 申请GPU内存
    float* dev_a = nullptr;
    float* dev_b = nullptr;
    float* dev_c = nullptr;
    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_b, N * sizeof(float));
    hipMalloc((void**)&dev_c, blocksPerGrid * sizeof(float));

    //将数据从主机copy进GPU
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

    //进行点积计算
    dot<<<32, 256>>>(dev_a, dev_b, dev_c);

    //将计算结果copy回主机
    hipMemcpy(c, dev_c, blocksPerGrid * sizeof(float), hipMemcpyDeviceToHost);

    //将每个block的结果进行累加
    for (size_t i = 1; i < blocksPerGrid; i++)
        c[0] += c[i];

    // 输出结果
    std::cout << "The ground truth is 16384, our answer is " << c[0] << std::endl;

    //释放内存
    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);

    system("pause");
    return 0;
}
